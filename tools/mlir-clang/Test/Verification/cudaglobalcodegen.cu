#include "hip/hip_runtime.h"
// RUN: mlir-clang %s --cuda-gpu-arch=sm_60 -nocudalib -nocudainc %resourcedir --function=* -S | FileCheck %s

#include <cstddef>

struct dim3
{
	unsigned int x, y, z;
	dim3(unsigned int vx = 1, unsigned int vy = 1, unsigned int vz = 1) : x(vx), y(vy), z(vz) {}

};


#define __global__ __attribute__((global))

extern "C" unsigned __cudaPushCallConfiguration(dim3 gridDim, dim3 blockDim,
                                                size_t sharedMem = 0,
                                                void *stream = 0);

__global__ void bar(int * a)
{
#ifdef __CUDA_ARCH__
    *a = 1;
#else
    *a = 2;
#endif
}

void baz(int * a){
    bar<<<dim3(1,1,1), dim3(1,1,1)>>>(a);
}
// CHECK:  func private @_Z18__device_stub__barPi(%arg0: memref<?xi32>)
// CHECK-NEXT:    %c1_i32 = arith.constant 1 : i32
// CHECK-NEXT:    affine.store %c1_i32, %arg0[0] : memref<?xi32>
// CHECK-NEXT:    return
// CHECK-NEXT:  }
// CHECK:  func @_Z3bazPi(%arg0: memref<?xi32>) attributes {llvm.linkage = #llvm.linkage<external>} {
// CHECK-NEXT:    %c1 = arith.constant 1 : index
// CHECK-NEXT:    gpu.launch blocks(%arg1, %arg2, %arg3) in (%arg7 = %c1, %arg8 = %c1, %arg9 = %c1) threads(%arg4, %arg5, %arg6) in (%arg10 = %c1, %arg11 = %c1, %arg12 = %c1) {
// CHECK-NEXT:      call @_Z18__device_stub__barPi(%arg0) : (memref<?xi32>) -> ()
// CHECK-NEXT:      gpu.terminator
// CHECK-NEXT:    }
// CHECK-NEXT:    return
// CHECK-NEXT:  }
